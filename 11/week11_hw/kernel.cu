#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

//1 exercise
//2 prac 1-3
//3 Fibonacci
//4 gen
//5 hw1
#define prac 2


#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1
#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

/*
if block size is 8,					/ gpu: 13.84512
if block size is 16,				/ gpu: 8.617824
if block size is 24,				/ gpu: 9.498592
if block size is 32, cpu: 79.871399 / gpu: 9.26400
*/

#if prac==1

typedef struct {
	int width;
	int height;
	float *elements;
} Array;


#define MAX_N_ELEMENTS	(1 << 25)

void generate_random_float_array(float *array, int n) {

	int i;

	for (i = 0; i < n; i++) {
		array[i] = 3.1415926f*((float)rand() / RAND_MAX);
	}
}
void combine_two_arrays(float *x, float *y, float *z, int n) {
	int i;

	for (i = 0; i < n; i++) {
		z[i] = 1.0f / (sin(x[i])*cos(y[i]) + cos(x[i])*sin(y[i]));
	}
}

__global__ void CombineTwoArrraysKernel(Array A, Array B, Array C) {
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	C.elements[id] = 1.0f / (sin(A.elements[id])*cos(B.elements[id])+ cos(A.elements[id])*sin(B.elements[id]));
}

hipError_t combine_two_arrays_GPU(const Array A, const Array B, Array C);

int BLOCK_SIZE = 32;

int main()
{
	int n_elements;

	srand((unsigned int)time(NULL));
	n_elements = MAX_N_ELEMENTS;
	Array A, B, C, G;
	A.width = B.width = C.width = G.width =1024;
	A.height = B.height = C.height = G.height = MAX_N_ELEMENTS / 1024;

	A.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
	B.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
	C.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
	G.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
	generate_random_float_array(A.elements, MAX_N_ELEMENTS);
	generate_random_float_array(B.elements, MAX_N_ELEMENTS);
	
	CHECK_TIME_START;
	combine_two_arrays(A.elements, B.elements, C.elements, n_elements);
	CHECK_TIME_END(compute_time);

	printf("***CPU C[10] = %f/ Time taken = %.6fms\n", C.elements[10], compute_time);

	//CHECK_TIME_START;
	hipError_t cudaStatus = combine_two_arrays_GPU(A, B, G);
	//CHECK_TIME_END(compute_time);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "combine_two_arrays_GPU failed!");
        return 1;
    }
	printf("***GPU G[10] = %f/ Time taken = %.6fms\n", G.elements[10], device_time);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
hipError_t combine_two_arrays_GPU(const Array A, const Array B, Array C) {
	
	//�Ʒ� �Լ����� ����Ͽ� ��� �ϸ� ������ ��Ȯ�� �ð��� ������ �� ������ �����غ� ��.
	CHECK_TIME_INIT_GPU()
	
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}/////////////  if(cu.....  ==CUDA_CALL
	
	Array d_A, d_B, d_C;
	size_t size;

	d_A.width = A.width; d_A.height = A.height;
	size = A.width * A.height * sizeof(float);
	CUDA_CALL(hipMalloc(&d_A.elements, size))
	
	CUDA_CALL(hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice))

	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	CUDA_CALL(hipMalloc(&d_B.elements, size))
	CUDA_CALL(hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice))

	d_C.width = C.width; d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	CUDA_CALL(hipMalloc(&d_C.elements, size))

	// Assume that width and height are multiples of BLOCK SIZE.
	
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(A.width / dimBlock.x, A.height / dimBlock.y);
	CHECK_TIME_START_GPU()
	CombineTwoArrraysKernel <<< dimGrid, dimBlock >>> (d_A, d_B, d_C);
	CHECK_TIME_END_GPU(device_time)

	CUDA_CALL(hipGetLastError())

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CUDA_CALL(hipDeviceSynchronize())
	CUDA_CALL(hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost))
	

	CHECK_TIME_DEST_GPU()

Error:
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	return cudaStatus;
}
#endif


//N = 4096
//CPU: 11.082584
//Block Size 32 -> 3.419040
//Block Size 24 -> 0.3448
//Block Size 16 -> 0.371936
//Block Size 8 -> 0.369760
#if prac==2
typedef struct {
	int width;
	int height;
	float *elements;
} Array;

int n;
#define BLOCK_SIZE 16
const int ELEM_PER_VECTOR = 32;
float (*pVecX)[ELEM_PER_VECTOR], (*pVecY)[ELEM_PER_VECTOR], (*pVecY_G)[ELEM_PER_VECTOR];
float(*pMatA)[ELEM_PER_VECTOR];

void init_MatVec(void)
{
	srand((unsigned)time(NULL));
	FILE* fp = fopen("gen.bin", "rb");
	fread(&n, sizeof(float), 1, fp);
	printf("n: %d\n", n);
	pVecX = new float[n][ELEM_PER_VECTOR];
	pVecY = new float[n][ELEM_PER_VECTOR];
	pVecY_G = new float[n][ELEM_PER_VECTOR];
	pMatA = new float[ELEM_PER_VECTOR][ELEM_PER_VECTOR];

	fread(pVecX, sizeof(float), n * ELEM_PER_VECTOR, fp);
	fread(pMatA, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);
	fclose(fp);
}
void Mat_Vec_Multiply()
{
	int i, j, k;
	float sum;
	for( k = 0 ; k < n; k++){
		for( i = 0 ; i < ELEM_PER_VECTOR ; i++){
			sum = 0;
			for(j = 0 ; j < ELEM_PER_VECTOR ; j++){
				sum += pMatA[i][j] * pVecX[k][j];
			}
			pVecY[k][i] = sum;
		}
	}
}

__global__ void Mat_Vec_Multiply_Kernel(const Array VecX, const Array MatA, Array VecY)
{ 
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int id = gridDim.x*blockDim.x*row + col;
	int i,j,k;


	VecY.elements[row*ELEM_PER_VECTOR + col] = 0;
	for(k = 0 ; k < ELEM_PER_VECTOR ; k++)
		VecY.elements[row*ELEM_PER_VECTOR + col] += (MatA.elements[col * ELEM_PER_VECTOR + k] * VecX.elements[row*ELEM_PER_VECTOR+k]);
	
}

void Mat_Vec_Multiply_GPU()
{
	//�Ʒ� �Լ����� ����Ͽ� ��� �ϸ� ������ ��Ȯ�� �ð��� ������ �� ������ �����غ� ��.
	CHECK_TIME_INIT_GPU()
	
	
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}/////////////  if(cu.....  ==CUDA_CALL
	
	Array _pVecX, _pMatA, _pVecY_G;
	size_t size;



 	_pVecX.width = ELEM_PER_VECTOR; _pVecX.height = n;
	size = n * ELEM_PER_VECTOR * sizeof(float);
	CUDA_CALL(hipMalloc(&_pVecX.elements, size))	
	CUDA_CALL(hipMemcpy(_pVecX.elements, pVecX, size, hipMemcpyHostToDevice))

	_pMatA.width = ELEM_PER_VECTOR; _pVecX.height = ELEM_PER_VECTOR;
	size = ELEM_PER_VECTOR * ELEM_PER_VECTOR * sizeof(float);
	CUDA_CALL(hipMalloc(&_pMatA.elements, size))	
	CUDA_CALL(hipMemcpy(_pMatA.elements, pMatA, size, hipMemcpyHostToDevice))

	_pVecY_G.width = ELEM_PER_VECTOR; _pVecY_G.height = n;
	size = n * ELEM_PER_VECTOR * sizeof(float);
	CUDA_CALL(hipMalloc(&_pVecY_G.elements, n*ELEM_PER_VECTOR))

	// Assume that width and height are multiples of BLOCK SIZE.
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid((ELEM_PER_VECTOR *ELEM_PER_VECTOR )/ dimBlock.x, n / dimBlock.y);
	dim3 dimGrid(_pVecX.width/ dimBlock.x, _pVecX.height / dimBlock.y); //32 n
	CHECK_TIME_START_GPU()
	Mat_Vec_Multiply_Kernel <<< dimGrid, dimBlock >>> (_pVecX, _pMatA, _pVecY_G);
	CHECK_TIME_END_GPU(device_time)
	CUDA_CALL(hipGetLastError())

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CUDA_CALL(hipDeviceSynchronize())


	CUDA_CALL(hipMemcpy(pVecY_G, _pVecY_G.elements, n*ELEM_PER_VECTOR, hipMemcpyDeviceToHost))
	

	CHECK_TIME_DEST_GPU()

Error:
	hipFree(_pVecX.elements);
	hipFree(_pMatA.elements);
	hipFree(_pVecY_G.elements);

}

int main()
{
	init_MatVec();
	printf("n = %d  file open ok.\n", n);

	CHECK_TIME_START;
	Mat_Vec_Multiply();
	CHECK_TIME_END(compute_time);
	printf("***CPU C[10] = %.3f/ Time taken = %.6fms\n", pVecY[0][0], compute_time);
	Mat_Vec_Multiply_GPU();
	printf("***GPU C[10] = %.3f/ Time taken = %.6fms\n", pVecY_G[0][0], device_time);
		
}


#endif

#if prac==3
#define BLOCK_SIZE 8
#define N 67108864 // 8192 * 8192 = 2^13 * 2^13

int Fibonacci(int n) {
	// DO NOT MODIFY THIS FUNCTION!!!
	float sqrt_5, x_0, x_1;
	float tmp_0, tmp_1;

	sqrt_5 = sqrtf(5.0f);
	x_0 = (1.0f + sqrt_5) / 2.0f;
	x_1 = (1.0f - sqrt_5) / 2.0f;

	tmp_0 = tmp_1 = 1.0f;
	for (int i = 0; i < n; i++) {
		tmp_0 *= x_0;
		tmp_1 *= x_1;
	}
	return (int)((tmp_0 - tmp_1) / sqrt_5 + 0.5);
}
void generate_input(int *x, int n) {
	// DO NOT MODIFY THIS FUNCTION!!!
	srand((unsigned int)time(NULL));

	for (int i = 0; i < n; i++) {
		x[i] = 35 + (int)(5.0f * rand() / RAND_MAX + 0.5f);
	}
}
__global__ void Fibonacci_Kernel(int *x, int *y)
{
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	//int id = threadIdx.x;
	float sqrt_5, x_0, x_1;
	float tmp_0, tmp_1;

	sqrt_5 = sqrtf(5.0f);
	x_0 = (1.0f + sqrt_5) / 2.0f;
	x_1 = (1.0f - sqrt_5) / 2.0f;

	tmp_0 = tmp_1 = 1.0f;
	for (int i = 0; i < x[id]; i++) {
		tmp_0 *= x_0;
		tmp_1 *= x_1;
	}
	y[id] = (int)((tmp_0 - tmp_1) / sqrt_5 + 0.5);
}
void Fibonacci_GPU(int *x, int *y)
{
	CHECK_TIME_INIT_GPU()
	hipError_t cudaStatus;
		// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	int *_y, *_x;
	size_t size = N * sizeof(int);
	size_t rtsize = (size_t)(sqrt((float)N));

	CUDA_CALL(hipMalloc(&_y, size))

	CUDA_CALL(hipMalloc(&_x, size))	
	CUDA_CALL(hipMemcpy(_x, x, size, hipMemcpyHostToDevice))	

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid( rtsize / dimBlock.x,  rtsize / dimBlock.y);
	dim3 dimGrid( 1024 / dimBlock.x,  (N/1024) / dimBlock.y);
	CHECK_TIME_START_GPU()
	Fibonacci_Kernel <<< dimGrid, dimBlock >>> (_x, _y);
	CHECK_TIME_END_GPU(device_time)
	CUDA_CALL(hipGetLastError())

	CUDA_CALL(hipDeviceSynchronize())

	CUDA_CALL(hipMemcpy(y, _y, size, hipMemcpyDeviceToHost))
	CHECK_TIME_DEST_GPU()

Error:
	hipFree(_x);
	hipFree(_y);

}

int *x, *y_c, *y_g; // input/output arrays

void main(void) {
	
	int n, i;
	FILE *fp;

	// Read the input array from the input file if one already exists.
	fp = fopen("x.binary", "rb");
	if (!fp) {
		fprintf(stderr, "Error: cannot open the input file...\n");
		exit(-1);
	}
	fread(&n, sizeof(int), 1, fp);
	fprintf(stdout, "\n*** The problem size is %d.\n", n);

	x = (int *)malloc(sizeof(int)*n);
	if (!x) {
		fprintf(stderr, "Error: cannot allocate memory for the input array...\n");
		exit(-1);
	}
	fread(x, sizeof(int), n, fp);
	fclose(fp);

	y_c = (int *)malloc(sizeof(int)*n); 
	y_g = (int *)malloc(sizeof(int)*n);

	//CPU
	
	CHECK_TIME_START;
	for (i = 0; i < n; i++) {
		y_c[i] = Fibonacci(x[i]);
	}
	CHECK_TIME_END(compute_time);
	fprintf(stdout, "\n***_CPU_ Time taken for computing %d Fibonacci numbers is %.6fms\n\n", n, compute_time);
	
	//GPU
	Fibonacci_GPU(x, y_g);

	fprintf(stdout, "\n***_GPU_ Time taken for computing %d Fibonacci numbers is %.6fms\n\n", n, device_time);

	i = (int)(n * (rand() / (RAND_MAX + 1.0f)));
	fprintf(stdout, "*** Fibonacci number of %d is (CPU :%d  , GPU :%d).\n\n", x[i], y_c[i], y_g[i]);

	// Write the output array into the output file.
	fp = fopen("y.binary", "wb");
	if (!fp) {
		fprintf(stderr, "Error: cannot open the output file...\n");
		exit(-1);
	}
	fwrite(&n, sizeof(int), 1, fp);
	fwrite(y_c, sizeof(int), n, fp);
	fclose(fp);

	free(x);
	free(y_c);
	free(y_g);
}


#endif

#if prac==4

const int ELEM_PER_VECTOR = 32;

int main()
{
	int n;

	srand((unsigned)&n);

	printf("Enter a size: ");
	scanf("%d", &n);

	int size = ELEM_PER_VECTOR * n;
	float* vec = new float[size];

	for (int i = 0; i < size; ++i)
	{
		vec[i] = (float(rand()) * 2.f / RAND_MAX) - 1.f;
	}

	float(*mat)[ELEM_PER_VECTOR] = new float[ELEM_PER_VECTOR][ELEM_PER_VECTOR];
	for (int i = 0; i < ELEM_PER_VECTOR; ++i)
	{
		for (int j = 0; j < ELEM_PER_VECTOR; ++j)
		{
			mat[i][j] = (float(rand()) * 2.f / RAND_MAX) - 1.f;
		}
	}

	FILE* fp = fopen("gen.bin", "wb");
	fwrite(&n, sizeof(float), 1, fp);
	fwrite(vec, sizeof(float), size, fp);
	fwrite(mat, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);
	fclose(fp);

	fp = fopen("gen.bin", "rb");
	float* vec2 = new float[size];
	float(*mat2)[ELEM_PER_VECTOR] = new float[ELEM_PER_VECTOR][ELEM_PER_VECTOR];
	int m;
	fread(&m, sizeof(float), 1, fp);
	fread(vec2, sizeof(float), m * ELEM_PER_VECTOR, fp);
	fread(mat2, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);

	if (n != m) printf("error: size diff. %n != %n", n, m);
	for (int i = 0; i < size; ++i)
	{
		if (vec[i] != vec2[i])
		{
			printf("[%d] %f != %f\n", vec[i], vec2[i]);
			break;
		}
	}
	for (int i = 0; i < ELEM_PER_VECTOR; ++i)
	{
		for (int j = 0; j < ELEM_PER_VECTOR; ++j)
		{
			if (mat[i][j] != mat2[i][j])
			{
				printf("[%d][%d]\n", i, j);
				break;
			}
		}
	}
	fclose(fp);

	delete[] vec;

	return 0;
}

#endif


#if prac==5

#define N_EQUATIONS 1048576
#define BLOCK_SIZE 16
float *A, *B, *C;

void find_roots_CPU(float *A, float *B, float *C, float *X0, float *X1, float *FX0, float *FX1, int n) 
{ 
	int i; 
	float a, b, c, d, x0, x1, tmp;

	for (i = 0; i < n; i++) {
		a = A[i]; b = B[i]; c = C[i]; 
		d = sqrtf(b*b - 4.0f*a*c); 
		tmp = 1.0f / (2.0f*a); 
		X0[i] = x0 = (-b - d) * tmp; 
		X1[i] = x1 = (-b + d) * tmp; 
		FX0[i] = (a*x0 + b)*x0 + c; 
		FX1[i] = (a*x1 + b)*x1 + c;
	}
}

__global__ void find_roots_Kernel(float *A, float *B, float *C, float *X0, float *X1, float *FX0, float *FX1, int n)
{ 
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int i = gridDim.x*blockDim.x*row + col;
	
	float a, b, c, d, x0, x1, tmp;

	a = A[i]; b = B[i]; c = C[i]; 
	d = sqrtf(b*b - 4.0f*a*c); 
	tmp = 1.0f / (2.0f*a); 
	X0[i] = x0 = (-b - d) * tmp; 
	X1[i] = x1 = (-b + d) * tmp; 
	FX0[i] = (a*x0 + b)*x0 + c; 
	FX1[i] = (a*x1 + b)*x1 + c;
}

void find_roots_GPU(float *A, float *B, float *C, float *X0, float *X1, float *FX0, float *FX1, int n)
{

	CHECK_TIME_INIT_GPU()
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}/////////////  if(cu.....  ==CUDA_CALL
	
	//Array _pVecX, _pMatA, _pVecY_G;
	float *_A, *_B, *_C;
	float *_X0, *_X1, *_FX0, *_FX1;
	size_t size = n*sizeof(float);
	size_t rtsize = (size_t)sqrt(float(n));


	CUDA_CALL(hipMalloc(&_A, size))	
	CUDA_CALL(hipMemcpy(_A, A, size, hipMemcpyHostToDevice))
	CUDA_CALL(hipMalloc(&_B, size))	
	CUDA_CALL(hipMemcpy(_B, B, size, hipMemcpyHostToDevice))
	CUDA_CALL(hipMalloc(&_C, size))	
	CUDA_CALL(hipMemcpy(_C, C, size, hipMemcpyHostToDevice))

	CUDA_CALL(hipMalloc(&_X0, size))
	CUDA_CALL(hipMalloc(&_X1, size))
	CUDA_CALL(hipMalloc(&_FX0, size))
	CUDA_CALL(hipMalloc(&_FX1, size))

	// Assume that width and height are multiples of BLOCK SIZE.
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid( rtsize/ dimBlock.x, rtsize / dimBlock.y);
	dim3 dimGrid( 512/ dimBlock.x, 2048 / dimBlock.y);

	CHECK_TIME_START_GPU()
	find_roots_Kernel <<< dimGrid, dimBlock >>> (_A, _B, _C, _X0, _X1, _FX0, _FX1, n);
	CHECK_TIME_END_GPU(device_time)
	CUDA_CALL(hipGetLastError())

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CUDA_CALL(hipDeviceSynchronize())
	

	CUDA_CALL(hipMemcpy(X0, _X0, size, hipMemcpyDeviceToHost))
	CUDA_CALL(hipMemcpy(X1, _X1, size, hipMemcpyDeviceToHost))
	CUDA_CALL(hipMemcpy(FX0, _FX0, size, hipMemcpyDeviceToHost))
	CUDA_CALL(hipMemcpy(FX1, _FX1, size, hipMemcpyDeviceToHost))

	
	CHECK_TIME_DEST_GPU()

Error:
	hipFree(_A);
	hipFree(_B);
	hipFree(_C);
	hipFree(_X0);
	hipFree(_X1);
	hipFree(_FX0);
	hipFree(_FX1);
}

void read_poly(){
	//float *_A, float *_B, float *_C
	int i, n = N_EQUATIONS;

	
	FILE *fA = fopen("A.bin", "rb");
	if(!fA){
		printf("file open error\n");
		exit(-1);
	}	
	fread(&n, sizeof(float), 1, fA);
	A = (float*)malloc(sizeof(float)*N_EQUATIONS);
	fread(A, sizeof(float), N_EQUATIONS, fA);
	fclose(fA);
	

	FILE *fB = fopen("B.bin", "rb");
	if(!fB){
		printf("file open error\n");
		exit(-1);
	}
	fread(&n, sizeof(float), 1, fB);
	B = (float*)malloc(sizeof(float)*N_EQUATIONS);
	fread(B, sizeof(float), n, fB);
	fclose(fB);

	FILE *fC = fopen("C.bin", "rb");
	if(!fC){
		printf("file open error\n");
		exit(-1);
	}
	fread(&n, sizeof(float), 1, fC);
	C = (float*)malloc(sizeof(float)*N_EQUATIONS);
	fread(C, sizeof(float), n, fC);
	fclose(fC);
}

void write_poly(float *X0, float *X1, float *FX0, float *FX1, int n)
{
	FILE *x0, *x1, *fx0, *fx1;
	size_t cnt;

	x0 = fopen("X0.bin", "wb");
	cnt = fwrite (X0, 4, n, x0);
	fclose(x0);

	x1 = fopen("X1.bin", "wb");
	cnt = fwrite (X1, 4, n, x1);
	fclose(x1);
	
	fx0 = fopen("FX0.bin", "wb");
	cnt = fwrite (FX0, 4, n, fx0);
	fclose(fx0);

	fx1 = fopen("FX1.bin", "wb");
	cnt = fwrite (FX1, 4, n, fx1);
	fclose(fx1);

}

int main(){
//	float *A, *B, *C;
	float *cX0, *cX1, *cFX0, *cFX1, *gX0, *gX1, *gFX0, *gFX1;
	int n = N_EQUATIONS;

	//read files
	read_poly();
	printf("n = %d  file open ok.\n", n);

	//check CPU time
	cX0 = (float*)malloc(sizeof(float)*n);
	cX1 = (float*)malloc(sizeof(float)*n);
	cFX0 = (float*)malloc(sizeof(float)*n);
	cFX1 = (float*)malloc(sizeof(float)*n);

	CHECK_TIME_START;
	find_roots_CPU(A,B,C, cX0, cX1, cFX0, cFX1, n);
	CHECK_TIME_END(compute_time);
	printf("***CPU Time taken = %.6fms\n", compute_time);
	
	//check GPU time
	gX0 = (float*)malloc(sizeof(float)*n);
	gX1 = (float*)malloc(sizeof(float)*n);
	gFX0 = (float*)malloc(sizeof(float)*n);
	gFX1 = (float*)malloc(sizeof(float)*n);

	find_roots_GPU(A,B,C, gX0, gX1, gFX0, gFX1, n);
	printf("***GPU Time taken = %.6fms\n", device_time);

	//check if same result
	printf("CPU result X0[1048575], fX0[1048575] = %f, %f\n", cX0[1048575], cFX0[1048575]);
	printf("GPU result X0[1048575], fX0[1048575] = %f, %f\n", gX0[1048575], gFX0[1048575]);

	//save in file
	write_poly(gX0, gX1, gFX0, gFX1, n);

}


#endif














